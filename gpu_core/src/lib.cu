#include "hip/hip_runtime.h"
#include <math.h>

#ifdef __NVCC__
#define gpuFree hipFree
#define gpuMalloc hipMalloc
#define gpuMemcpy hipMemcpy
#define gpuMemcpyHostToDevice hipMemcpyHostToDevice
#define gpuMemcpyDeviceToHost hipMemcpyDeviceToHost
#define gpuMemcpyDeviceToDevice hipMemcpyDeviceToDevice

#else
#include <hip/hip_runtime.h>
#define gpuFree hipFree
#define gpuMalloc hipMalloc
#define gpuMemcpy hipMemcpy
#define gpuMemcpyHostToDevice hipMemcpyHostToDevice
#define gpuMemcpyDeviceToHost hipMemcpyDeviceToHost
#define gpuMemcpyDeviceToDevice hipMemcpyDeviceToDevice
#endif

typedef unsigned long ulong;

extern "C" void *gpu_malloc(ulong size)
{
    void *ptr;
    gpuMalloc(&ptr, size);
    return ptr;
}

extern "C" void gpu_free(void *ptr)
{
    gpuFree(ptr);
}

extern "C" void gpu_memcpy_htod(void *dst, const void *src, ulong size)
{
    gpuMemcpy(dst, src, size, gpuMemcpyHostToDevice);
}

extern "C" void gpu_memcpy_dtoh(void *dst, const void *src, ulong size)
{
    gpuMemcpy(dst, src, size, gpuMemcpyDeviceToHost);
}

extern "C" void gpu_memcpy_dtod(void *dst, const void *src, ulong size)
{
    gpuMemcpy(dst, src, size, gpuMemcpyDeviceToDevice);
}

extern "C" void gpu_device_synchronize()
{
#ifdef __NVCC__
    hipDeviceSynchronize();
#else
    hipDeviceSynchronize();
#endif
}

extern "C" int gpu_get_device_count()
{
    int count;
#ifdef __NVCC__
    hipGetDeviceCount(&count);
#else
    hipGetDeviceCount(&count);
#endif
    return count;
}

extern "C" int gpu_get_device()
{
    int device;
#ifdef __NVCC__
    hipGetDevice(&device);
#else
    hipGetDevice(&device);
#endif
    return device;
}

extern "C" void gpu_set_device(int device)
{
#ifdef __NVCC__
    hipSetDevice(device);
#else
    hipSetDevice(device);
#endif
}

// Adapted from:
// https://sodocumentation.net/cuda/topic/6566/parallel-reduction--e-g--how-to-sum-an-array

#define REDUCE_BLOCK_SIZE 1024
#define REDUCE_GRID_SIZE 24

static __global__ void vec_max_f64_kernel(const double *in, ulong N, double *out)
{
    __shared__ double lds[REDUCE_BLOCK_SIZE];

    ulong start = threadIdx.x + blockIdx.x * REDUCE_BLOCK_SIZE;
    ulong gsize = gridDim.x * REDUCE_BLOCK_SIZE;
    double max = in[0];

    for (ulong i = start; i < N; i += gsize)
    {
        max = fmax(max, in[i]);
    }
    lds[threadIdx.x] = max;

    __syncthreads();

    for (ulong size = REDUCE_BLOCK_SIZE / 2; size > 0; size /= 2)
    {
        if (threadIdx.x < size)
        {
            lds[threadIdx.x] = fmax(lds[threadIdx.x], lds[threadIdx.x + size]);
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
    {
        out[blockIdx.x] = lds[0];
    }
}

extern "C" void gpu_vec_max_f64(const double *vec, ulong size, double *result)
{
    if (size == 0) {
        return;
    }
    double* block_max;
    gpuMalloc(&block_max, sizeof(double) * REDUCE_GRID_SIZE);

    vec_max_f64_kernel<<<REDUCE_GRID_SIZE, REDUCE_BLOCK_SIZE>>>(vec, size, block_max);
    vec_max_f64_kernel<<<1, REDUCE_BLOCK_SIZE>>>(block_max, REDUCE_GRID_SIZE, block_max);

    gpuMemcpy(result, block_max, sizeof(double), gpuMemcpyDeviceToDevice);
    gpuFree(block_max);
}
