
#include <hip/hip_runtime.h>
extern "C" void *gpu_malloc(unsigned long size) {
    void *ptr;
    hipMalloc(&ptr, size);
    return ptr;
}

extern "C" void gpu_free(void *ptr) {
    hipFree(&ptr);
}

extern "C" void gpu_memcpy_htod(void *dst, const void *src, unsigned long size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

extern "C" void gpu_memcpy_dtoh(void *dst, const void *src, unsigned long size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

extern "C" void gpu_memcpy_dtod(void *dst, const void *src, unsigned long size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
}
